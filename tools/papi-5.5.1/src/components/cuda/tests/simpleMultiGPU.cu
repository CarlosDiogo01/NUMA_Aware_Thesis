#include "hip/hip_runtime.h"
/* PAPI Multiple GPU example.  This example is taken from the NVIDIA
 * documentation (Copyright 1993-2013 NVIDIA Corporation) and has been
 * adapted to show the use of CUPTI and PAPI in collecting event
 * counters for multiple GPU contexts.  PAPI Team (2015)
 */

/*
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This application demonstrates how to use the CUDA API to use multiple GPUs,
 * with an emphasis on simple illustration of the techniques (not on performance).
 *
 * Note that in order to detect multiple GPUs in your system you have to disable
 * SLI in the nvidia control panel. Otherwise only one GPU is visible to the
 * application. On the other side, you can still extend your desktop to screens
 * attached to both GPUs.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cupti.h>
#include <timer.h>

#include "papi_test.h"

#if not defined PAPI
#undef PAPI
#endif

#if not defined CUPTI_ONLY
#undef CUPTI_ONLY
#endif

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

#include "simpleMultiGPU.h"

// //////////////////////////////////////////////////////////////////////////////
// Data configuration
// //////////////////////////////////////////////////////////////////////////////
const int MAX_GPU_COUNT = 32;
const int DATA_N = 48576 * 32;
#ifdef PAPI
const int MAX_NUM_EVENTS = 32;
#endif

#define CHECK_CU_ERROR(err, cufunc)                                     \
    if (err != hipSuccess) { printf ("Error %d for CUDA Driver API function '%s'\n", err, cufunc); return -1; }

#define CHECK_CUDA_ERROR(err)                                           \
    if (err != hipSuccess) { printf ("Error %d for CUDA \n", err ); return -1; }

#define CHECK_CUPTI_ERROR(err, cuptifunc)                               \
    if (err != CUPTI_SUCCESS) { printf ("Error %d for CUPTI API function '%s'\n", err, cuptifunc); return -1; }


// //////////////////////////////////////////////////////////////////////////////
// Simple reduction kernel.
// Refer to the 'reduction' CUDA SDK sample describing
// reduction optimization strategies
// //////////////////////////////////////////////////////////////////////////////
__global__ static void reduceKernel( float *d_Result, float *d_Input, int N )
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int threadN = gridDim.x * blockDim.x;
    float sum = 0;
    
    for( int pos = tid; pos < N; pos += threadN )
        sum += d_Input[pos];
    
    d_Result[tid] = sum;
}

// //////////////////////////////////////////////////////////////////////////////
// Program main
// //////////////////////////////////////////////////////////////////////////////
int main( int argc, char **argv )
{
    // Solver config
    TGPUplan plan[MAX_GPU_COUNT];
    // GPU reduction results
    float h_SumGPU[MAX_GPU_COUNT];
    float sumGPU;
    double sumCPU, diff;
    int i, j, gpuBase, GPU_N;
    
    const int BLOCK_N = 32;
    const int THREAD_N = 256;
    const int ACCUM_N = BLOCK_N * THREAD_N;
    
    printf( "Starting simpleMultiGPU\n" );
    
    // Report on the available CUDA devices
    int computeCapabilityMajor = 0, computeCapabilityMinor = 0;
    int runtimeVersion = 0, driverVersion = 0;
    int deviceNum = -1;
    char deviceName[32];
    hipDevice_t dev;
    CHECK_CUDA_ERROR( hipGetDeviceCount( &GPU_N ) );
    if( GPU_N > MAX_GPU_COUNT ) GPU_N = MAX_GPU_COUNT;
    printf( "CUDA-capable device count: %i\n", GPU_N );
    for ( deviceNum=0; deviceNum<GPU_N; deviceNum++ ) {
        CHECK_CU_ERROR( hipDeviceGet( &dev, deviceNum ), "hipDeviceGet" );
        CHECK_CU_ERROR( hipDeviceGetName( deviceName, 32, dev ), "hipDeviceGetName" );
        CHECK_CU_ERROR( hipDeviceComputeCapability( &computeCapabilityMajor, &computeCapabilityMinor,  dev ), "hipDeviceComputeCapability" );
        hipRuntimeGetVersion( &runtimeVersion );
        hipDriverGetVersion( &driverVersion );
        printf( "CUDA Device %d: %s : computeCapability %d.%d runtimeVersion %d.%d driverVersion %d.%d\n", deviceNum, deviceName, computeCapabilityMajor, computeCapabilityMinor, runtimeVersion/1000, (runtimeVersion%100)/10, driverVersion/1000, (driverVersion%100)/10 );
        if ( computeCapabilityMajor < 2 ) {
            printf( "CUDA Device %d compute capability is too low... will not add any more GPUs\n", deviceNum );
            GPU_N = deviceNum;
            break;
        }
    }
    uint32_t cupti_linked_version;
    cuptiGetVersion( &cupti_linked_version );
    printf("CUPTI version: Compiled against version %d; Linked against version %d\n", CUPTI_API_VERSION, cupti_linked_version );
    
    printf( "Generating input data...\n" );
    
    // Subdividing input data across GPUs
    // Get data sizes for each GPU
    for( i = 0; i < GPU_N; i++ )
        plan[i].dataN = DATA_N / GPU_N;
    // Take into account "odd" data sizes
    for( i = 0; i < DATA_N % GPU_N; i++ )
        plan[i].dataN++;
    
    // Assign data ranges to GPUs
    gpuBase = 0;
    for( i = 0; i < GPU_N; i++ ) {
        plan[i].h_Sum = h_SumGPU + i; // point within h_SumGPU array
        gpuBase += plan[i].dataN;
    }
    
    // Create streams for issuing GPU command asynchronously and allocate memory (GPU and System page-locked)
    for( i = 0; i < GPU_N; i++ ) {
        CHECK_CUDA_ERROR( hipSetDevice( i ) );
        // hipFree: forces creation of a context
        CHECK_CUDA_ERROR( hipFree( NULL ) );
        CHECK_CUDA_ERROR( hipStreamCreate( &plan[i].stream ) );
        // Allocate memory
        CHECK_CUDA_ERROR( hipMalloc( ( void ** ) &plan[i].d_Data, plan[i].dataN * sizeof( float ) ) );
        CHECK_CUDA_ERROR( hipMalloc( ( void ** ) &plan[i].d_Sum, ACCUM_N * sizeof( float ) ) );
        CHECK_CUDA_ERROR( hipHostMalloc( ( void ** ) &plan[i].h_Sum_from_device, ACCUM_N * sizeof( float ) ) );
        CHECK_CUDA_ERROR( hipHostMalloc( ( void ** ) &plan[i].h_Data, plan[i].dataN * sizeof( float ) ) );
        
        for( j = 0; j < plan[i].dataN; j++ ) {
            plan[i].h_Data[j] = ( float ) rand() / ( float ) RAND_MAX;
        }
    }
    
    
#ifdef CUPTI_ONLY
    printf("Setup CUPTI counters internally for elapsed_cycles_sm event (CUPTI_ONLY)\n");
    hipDevice_t device[MAX_GPU_COUNT];
    hipCtx_t ctx[MAX_GPU_COUNT];
    hipCtx_t ctxpopped[MAX_GPU_COUNT];
    CUpti_EventGroup eg[MAX_GPU_COUNT];
    CUpti_EventID myevent;//elapsed cycles
    for ( i=0; i<GPU_N; i++ ) {
        CHECK_CUDA_ERROR( hipSetDevice( i ) );
        CHECK_CU_ERROR( hipDeviceGet( &device[i], i ), "hipDeviceGet" );
        CHECK_CU_ERROR( hipCtxCreate( &ctx[i], 0, device[i] ), "hipCtxCreate" );
        CHECK_CUPTI_ERROR( cuptiEventGroupCreate( ctx[i], &eg[i], 0 ), "cuptiEventGroupCreate" );
        cuptiEventGetIdFromName ( device[i], "elapsed_cycles_sm", &myevent );
        CHECK_CUPTI_ERROR( cuptiEventGroupAddEvent( eg[i], myevent ), "cuptiEventGroupAddEvent" );
        CHECK_CUPTI_ERROR( cuptiEventGroupEnable( eg[i] ), "cuptiEventGroupEnable" );
        CHECK_CU_ERROR( hipCtxPopCurrent( &ctxpopped[i] ), "hipCtxPopCurrent" );
    }
#endif
    
#ifdef PAPI
    printf("Setup PAPI counters internally (PAPI)\n");
    int EventSet = PAPI_NULL;
    int NUM_EVENTS = MAX_GPU_COUNT*MAX_NUM_EVENTS;
    long long values[NUM_EVENTS];
    int eventCount;
    int retval, gg, ee;
    
    /* PAPI Initialization */
    retval = PAPI_library_init( PAPI_VER_CURRENT );
    if( retval != PAPI_VER_CURRENT ) fprintf( stderr, "PAPI_library_init failed\n" );
    printf( "PAPI version: %d.%d.%d\n", PAPI_VERSION_MAJOR( PAPI_VERSION ), PAPI_VERSION_MINOR( PAPI_VERSION ), PAPI_VERSION_REVISION( PAPI_VERSION ) );
    
    retval = PAPI_create_eventset( &EventSet );
    if( retval != PAPI_OK ) fprintf( stderr, "PAPI_create_eventset failed\n" );
    
    // In this example measure 2 events from each GPU
    int numEventEndings = 2;
    static char *EventEndings[] = { (char*)"inst_executed", (char *)"elapsed_cycles_sm" };
    
    // Add events at a GPU specific level ... eg cuda:::device:2:elapsed_cycles_sm
    char *EventName[NUM_EVENTS];
    char tmpEventName[50];
    eventCount = 0;
    for( gg = 0; gg < GPU_N; gg++ ) {
        CHECK_CUDA_ERROR( hipSetDevice( gg ) );         // Set device
        for ( ee=0; ee<numEventEndings; ee++ ) {
            snprintf( tmpEventName, 50, "cuda:::device:%d:%s\0", gg, EventEndings[ee] );
            printf( "Trying to add event %s to GPU %d in PAPI...", tmpEventName , gg );
            retval = PAPI_add_named_event( EventSet, tmpEventName );
            if (retval==PAPI_OK) {
                printf( "Added event\n" );
                EventName[eventCount] = (char *)calloc( 50, sizeof(char) );
                snprintf( EventName[eventCount], 50, "%s", tmpEventName );
                eventCount++;
            } else {
                printf( "Could not add event\n" );
            }
        }
    }
    
    // Start PAPI event measurement
    retval = PAPI_start( EventSet );
    if( retval != PAPI_OK )  fprintf( stderr, "PAPI_start failed\n" );
#endif
    
    // Start timing and compute on GPU(s)
    printf( "Computing with %d GPUs...\n", GPU_N );
    StartTimer();
    
    // Copy data to GPU, launch the kernel and copy data back. All asynchronously
    for( i = GPU_N-1; i >= 0; i-- ) {
        // Set device
        CHECK_CUDA_ERROR( hipSetDevice( i ) );
        //AYK CHECK_CUPTI_ERROR( cuptiEventGroupResetAllEvents ( eg[i] ), "cuptiEventGroupResetAllEvents" );
        // Copy input data from CPU
        CHECK_CUDA_ERROR( hipMemcpyAsync( plan[i].d_Data, plan[i].h_Data, plan[i].dataN * sizeof( float ), hipMemcpyHostToDevice, plan[i].stream ) );
        // Perform GPU computations
        reduceKernel <<< BLOCK_N, THREAD_N, 0, plan[i].stream >>> ( plan[i].d_Sum, plan[i].d_Data, plan[i].dataN );
        if ( hipGetLastError() != hipSuccess ) { printf( "reduceKernel() execution failed (GPU %d).\n", i ); exit(EXIT_FAILURE); }
        // Read back GPU results
        CHECK_CUDA_ERROR( hipMemcpyAsync( plan[i].h_Sum_from_device, plan[i].d_Sum, ACCUM_N * sizeof( float ), hipMemcpyDeviceToHost, plan[i].stream ) );
    }
    
    // Process GPU results
    printf( "Process GPU results on %d GPUs...\n", GPU_N );
    for( i = 0; i < GPU_N; i++ ) {
        float sum;
        // Set device
        CHECK_CUDA_ERROR( hipSetDevice( i ) );
        // Wait for all operations to finish
        hipStreamSynchronize( plan[i].stream );
        // Finalize GPU reduction for current subvector
        sum = 0;
        for( j = 0; j < ACCUM_N; j++ ) {
            sum += plan[i].h_Sum_from_device[j];
        }
        *( plan[i].h_Sum ) = ( float ) sum;
    }
    double gpuTime = GetTimer();


#ifdef CUPTI_ONLY
    size_t size = 1024;
    uint64_t buffer[1024];
    for ( i=0; i<GPU_N; i++ ) {
        CHECK_CUDA_ERROR( hipSetDevice( i ) );
        CHECK_CU_ERROR( hipCtxSynchronize( ), "hipCtxSynchronize" );
        CHECK_CUPTI_ERROR( cuptiEventGroupReadEvent ( eg[i], CUPTI_EVENT_READ_FLAG_NONE, myevent, &size, &buffer[i] ), "cuptiEventGroupReadEvent" );
        printf( "CUPTI elapsed_cycles_sm device %d counterValue %u\n", i, buffer[i] );
    }
#endif

#ifdef PAPI
    retval = PAPI_stop( EventSet, values );
    if( retval != PAPI_OK )  fprintf( stderr, "PAPI_stop failed\n" );
    for( i = 0; i < eventCount; i++ )
        printf( "PAPI counterValue %12lld \t\t --> %s \n", values[i], EventName[i] );
    retval = PAPI_cleanup_eventset( EventSet );
    if( retval != PAPI_OK )  fprintf( stderr, "PAPI_cleanup_eventset failed\n" );
    retval = PAPI_destroy_eventset( &EventSet );
    if( retval != PAPI_OK ) fprintf( stderr, "PAPI_destroy_eventset failed\n" );
    PAPI_shutdown();
#endif

    for( i = 0; i < GPU_N; i++ ) {
        CHECK_CUDA_ERROR( hipHostFree( plan[i].h_Sum_from_device ) );
        CHECK_CUDA_ERROR( hipFree( plan[i].d_Sum ) );
        CHECK_CUDA_ERROR( hipFree( plan[i].d_Data ) );
        // Shut down this GPU
        CHECK_CUDA_ERROR( hipStreamDestroy( plan[i].stream ) );
    }
    sumGPU = 0;
    for( i = 0; i < GPU_N; i++ ) {
        sumGPU += h_SumGPU[i];
    }
    printf( "  GPU Processing time: %f (ms)\n", gpuTime );

    // Compute on Host CPU
    printf( "Computing the same result with Host CPU...\n" );
    StartTimer();
    sumCPU = 0;
    for( i = 0; i < GPU_N; i++ ) {
        for( j = 0; j < plan[i].dataN; j++ ) {
            sumCPU += plan[i].h_Data[j];
        }
    }
    double cpuTime = GetTimer();
    printf( "  CPU Processing time: %f (ms)\n", cpuTime );

    // Compare GPU and CPU results
    printf( "Comparing GPU and Host CPU results...\n" );
    diff = fabs( sumCPU - sumGPU ) / fabs( sumCPU );
    printf( "  GPU sum: %f\n  CPU sum: %f\n", sumGPU, sumCPU );
    printf( "  Relative difference: %E \n", diff );

    // Cleanup and shutdown
    for( i = 0; i < GPU_N; i++ ) {
        CHECK_CUDA_ERROR( hipSetDevice( i ) );
        CHECK_CUDA_ERROR( hipHostFree( plan[i].h_Data ) );
        hipDeviceReset();
    }

    exit( ( diff < 1e-5 ) ? EXIT_SUCCESS : EXIT_FAILURE );
}

